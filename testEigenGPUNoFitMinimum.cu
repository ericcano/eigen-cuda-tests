#include "hip/hip_runtime.h"
#include <iostream>

#include <Eigen/Core>
#include <Eigen/Eigenvalues>
#include <random>

#include "Eigen/src/Core/Matrix.h"

//#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
//#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"
//#include "test_common.h"

#define TEST_DEBUG 1

using namespace Eigen;
using Matrix5d = Matrix<double, 5, 5>;


template <class C>
__host__ __device__ void printIt(C* m) {
#ifdef TEST_DEBUG
  printf("\nMatrix %dx%d\n", (int)m->rows(), (int)m->cols());
  for (u_int r = 0; r < m->rows(); ++r) {
    for (u_int c = 0; c < m->cols(); ++c) {
      printf("Matrix(%d,%d) = %f\n", r, c, (*m)(r, c));
    }
  }
#endif
}

template <class C1, class C2>
bool isEqualFuzzy(C1 a, C2 b, double epsilon = 1e-6) {
  for (unsigned int i = 0; i < a.rows(); ++i) {
    for (unsigned int j = 0; j < a.cols(); ++j) {
     if (std::abs(a(i, j) - b(i, j)) >= std::min(std::abs(a(i, j)), std::abs(b(i, j))) * epsilon) {
       printf("Failing in isEqualFuzzy: i=%d/%ld, j=%d/%ld, a(i,j)=%f, b(i,j)=%f, epsilon=%f\n", i, a.rows(), j, a.cols(), a(i,j), b(i,j), epsilon);
      }
      assert(std::abs(a(i, j) - b(i, j)) < std::min(std::abs(a(i, j)), std::abs(b(i, j))) * epsilon);
    }
  }
  return true;
}

bool isEqualFuzzy(double a, double b, double epsilon = 1e-6) {
  return std::abs(a - b) < std::min(std::abs(a), std::abs(b)) * epsilon;
}

template <typename T>
void fillMatrix(T& t) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(0.0, 2.0);
  for (int row = 0; row < t.rows(); ++row) {
    for (int col = 0; col < t.cols(); ++col) {
      t(row, col) = dis(gen);
    }
  }
  return;
}

#define cudaCheck(A) assert(hipSuccess == (A))

__global__ void kernelInverse4x4(Matrix4d *in, Matrix4d *out) { (*out) = in->inverse(); }

//__global__ void kernelInverse5x5(Matrix5d *in, Matrix5d *out) { (*out) = in->inverse(); }
__global__ void kernelInverse5x5(Matrix5d *in, Matrix5d *out) { (*out) = MatrixXd(*in).inverse(); }
//__global__ void kernelInverse5x5(Matrix5d *in, Matrix5d *out) { (*out) = Map<MatrixXd>(in->data(), 5, 5).inverse(); }

void testInverse4x4() {
  std::cout << "TEST INVERSE 4x4" << std::endl;
  Matrix4d m;
  fillMatrix(m);
  m += m.transpose().eval();

  Matrix4d m_inv = m.inverse();
  Matrix4d *mGPU = nullptr;
  Matrix4d *mGPUret = nullptr;
  Matrix4d *mCPUret = new Matrix4d();

#if TEST_DEBUG
  std::cout << "Here is the matrix m:" << std::endl << m << std::endl;
  std::cout << "Its inverse is:" << std::endl << m.inverse() << std::endl;
#endif
  cudaCheck(hipMalloc((void **)&mGPU, sizeof(Matrix4d)));
  cudaCheck(hipMalloc((void **)&mGPUret, sizeof(Matrix4d)));
  cudaCheck(hipMemcpy(mGPU, &m, sizeof(Matrix4d), hipMemcpyHostToDevice));

  kernelInverse4x4<<<1, 1>>>(mGPU, mGPUret);
  hipDeviceSynchronize();

  cudaCheck(hipMemcpy(mCPUret, mGPUret, sizeof(Matrix4d), hipMemcpyDeviceToHost));
#if TEST_DEBUG
  std::cout << "Its GPU inverse is:" << std::endl << (*mCPUret) << std::endl;
#endif
  assert(isEqualFuzzy(m_inv, *mCPUret));
}

void testInverse5x5() {
  std::cout << "TEST INVERSE 5x5" << std::endl;
  Matrix5d m;
  fillMatrix(m);
  m += m.transpose().eval();

  Matrix5d m_inv = m.inverse();
  Matrix5d *mGPU = nullptr;
  Matrix5d *mGPUret = nullptr;
  Matrix5d *mCPUret = new Matrix5d();

#if TEST_DEBUG
  std::cout << "Here is the matrix m:" << std::endl << m << std::endl;
  std::cout << "Its inverse is:" << std::endl << m.inverse() << std::endl;
#endif
  cudaCheck(hipMalloc((void **)&mGPU, sizeof(Matrix5d)));
  cudaCheck(hipMalloc((void **)&mGPUret, sizeof(Matrix5d)));
  cudaCheck(hipMemcpy(mGPU, &m, sizeof(Matrix5d), hipMemcpyHostToDevice));

  kernelInverse5x5<<<1, 1>>>(mGPU, mGPUret);
  hipDeviceSynchronize();

  cudaCheck(hipMemcpy(mCPUret, mGPUret, sizeof(Matrix5d), hipMemcpyDeviceToHost));
#if TEST_DEBUG
  std::cout << "Its GPU inverse is:" << std::endl << (*mCPUret) << std::endl;
#endif
  assert(isEqualFuzzy(m_inv, *mCPUret));
}


int main(int argc, char *argv[]) {
  testInverse4x4();
  testInverse5x5();
  return 0;
}
